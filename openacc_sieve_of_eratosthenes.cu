#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <openacc.h>

void sieveOfEratosthenes(int n) {
    // Step 1: Create a boolean array "prime[0..n]" and initialize all entries as true.
    std::vector<bool> prime(n + 1, true);

    // 0 and 1 are not prime numbers
    prime[0] = prime[1] = false;

    int sqrt_n = static_cast<int>(sqrt(n));

    // Step 2: Use OpenACC to parallelize the outer loop over primes
    #pragma acc data copy(prime[0:n+1])
    {
        for (int p = 2; p <= sqrt_n; p++) {
            if (prime[p]) {
                // Step 3: Parallelize the inner loop to mark multiples of p as false
                #pragma acc parallel loop
                for (int i = p * p; i <= n; i += p) {
                    prime[i] = false;
                }
            }
        }
    }

    // Step 4: Print all prime numbers
    std::cout << "Prime numbers up to " << n << " are: ";
    for (int i = 2; i <= n; i++) {
        if (prime[i]) {
            std::cout << i << " ";
        }
    }
    std::cout << std::endl;
}

int main() {
    int n;
    std::cout << "Enter the limit: ";
    std::cin >> n;

    sieveOfEratosthenes(n);

    return 0;
}
